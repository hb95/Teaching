
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>

struct State
{
    double x, v;
};

__host__ __device__ int size( State const& ){ return 2; }

template<typename F> __device__ State map(F f, State const& s){ return State{ f(s.x), f(s.v) }; }
template<typename F> __device__ State zip(F f, State const& s, State const& s2){ return State{ f(s.x, s2.x), f(s.v, s2.v) }; }
template<typename F> __device__ auto reducel(F f, State const& s){ return f(s.x, s.v); }

struct RungeKutta4Stepper
{
    double atol, rtol;

    template<typename TTime, typename TStep, typename TState, typename TRHS> __device__ 
    TState operator()( TTime time, TStep h, TState state, TRHS rhs, double* err) const
    {
        const auto scl_mul  = []__device__ (auto scl){ return [=]__device__ (auto x){ return scl*x; }; };
        const auto add      = []__device__ (auto const& x, auto const& y) { return x + y; };
        const auto rel_diff = [&]__device__ (auto const& x, auto const& y)
        {
           auto scale = atol + rtol * max(x, y);
           return (x-y)*(x-y)/scale/scale;
        };

        TState k1 = rhs(time, state);
        TState k2 = rhs(time + 0.5 * h, zip(add, state, map( scl_mul(h*0.5), k1 )));
        TState k3 = rhs(time + 0.5 * h, zip(add, state, map( scl_mul(h*0.5), k2 )));
	TState k4 = rhs(time + h,       zip(add, state, map( scl_mul(h),     k3 )));
	
        TState sum_state = zip(add, zip(add, k1, k4), map( scl_mul(2.0), zip(add, k2, k3)));
	TState res1 = zip(add, state, map( scl_mul(h/6.0), sum_state ) );//RK4 step
        TState res0 = zip(add, state, map( scl_mul(h), k1 ) );//Euler step

        *err = sqrt(reducel(add, zip(rel_diff, res1, res0)) / size(state));
        return res1;
    }
};

template<typename Stepper, typename T, typename H, typename RHS, typename S>
__global__ void step_impl(Stepper stepper, T t, int max_steps, H h0, RHS rhs, S* src, S* path, T* path0)
{
    auto i = blockIdx.x*blockDim.x+threadIdx.x;
    S s[2];
    int idx = 0;
    s[idx] = src[i];
    T time = t;
    H h = h0;

    double err = 1.0, lerr = 1e-4;

    int step = 0;
    while(step < max_steps)
    {
        do{
	    s[1-idx] = stepper(time, h, s[idx], rhs, &err);
            //if(i == 128 && err > 0.9){ printf("R %e %e %e\n", time, h, err); }
            
	    h = 0.95 * h * pow(err, -(1.0/4.0 - 0.75*0.4/4.0)) * pow(lerr, 0.4/4.0);         
            lerr = err;
        }while(err > 0.9);

        //if(i == 128){ printf("A %e %e %e\n", time, h, err); }	
        //__syncthreads();
	path[step * blockDim.x * gridDim.x + i] = s[1-idx];
	path0[step * blockDim.x * gridDim.x + i] = time;
	step += 1;
        idx = 1 - idx;
        time = time + h;
    }
}

template<typename Stepper, typename T, typename H, typename S, typename RHS>
std::pair<std::vector<T>, std::vector<S>> step(Stepper stepper, T t, int max_steps, H h, std::vector<S>const& src, RHS rhs)
{
    size_t n = src.size();
    static const size_t blockSize = 256;
           const size_t gridSize  = (size_t)ceil((float)n/blockSize);
    std::vector<S> resS(n*max_steps);
    std::vector<T> resT(n*max_steps);

    S* d_src;
    S* d_resS;
    T* d_resT;

    // Allocate memory for each vector on GPU
    hipMalloc(&d_src,  n*sizeof(S));
    hipMalloc(&d_resS, n*sizeof(S)*max_steps);
    hipMalloc(&d_resT, n*sizeof(T)*max_steps);

    // Copy host vectors to device
    hipMemcpy( d_src, src.data(), n*sizeof(S), hipMemcpyHostToDevice);

    //Measure time:
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    step_impl<<<gridSize, blockSize>>>(stepper, t, max_steps, h, rhs, d_src, d_resS, d_resT);
    hipEventRecord(stop);    
    
    hipEventSynchronize(stop);    
    float cuda_time = 0.0f;//msec
    hipEventElapsedTime(&cuda_time, start, stop);
    std::cout << "Elapsed time is: " << cuda_time << " msec\n";
    
    hipMemcpy( resS.data(), d_resS, n*sizeof(S)*max_steps, hipMemcpyDeviceToHost );
    hipMemcpy( resT.data(), d_resT, n*sizeof(T)*max_steps, hipMemcpyDeviceToHost );

    hipFree(d_src);
    hipFree(d_resS);
    hipFree(d_resT);
    
    return std::make_pair(resT, resS);	
}

int main()
{
	//using State = State;

	//Van der Pol oscillator
	double mu = 4.5;

	RungeKutta4Stepper rk4{5e-3, 5e-3};

	// Size of vectors
	size_t n0 = 64;
	size_t n = n0*n0;

	// State vectors
	std::vector<State> initial_state(n);

	// Initialize vectors on host
	for(int i = 0; i < n0; i++ )
	{
	    for(int j = 0; j < n0; j++ )
            {
               initial_state[i*n0+j].x = i*4.0/(n0-1)-2.0;
               initial_state[i*n0+j].v = j*4.0/(n0-1)-2.0;
	    }
	}

	auto rhs = [=]__device__ (double t, State const& s)
        {
             return State{ s.v, -mu*(s.x*s.x-1.0)*s.v - s.x };
        };

	auto res = step(rk4, 0.0, 2048*4, 1e-1, initial_state, rhs);

        {
          auto N = res.first.size() / n;
	  std::ofstream file("VdP.txt");
	  for(decltype(N) i=0; i<N; i++)
 	  {
            file << res.first[i*n+128] << "   " << res.second[i*n+128].x << "   " << res.second[i*n+128].v << "\n";
          }
		//std::cout << "result[" << i << "] = " << res[i].rabbits << ", " << res[i].wolves << "\n";
	}

	return 0;
}
